#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <>
#include "08_ParallelVectorAdd.cuh"

__global__ void vectorAdditionKernel(int *A, int *B, int *C, int arraySize)
{
    // Get thread ID.
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;

    // Check if thread is within array bounds.
    if (threadID < arraySize)
    {
        // Add a and b.
        C[threadID] = A[threadID] + B[threadID];
    }
}

/**
 * Wrapper function for the CUDA kernel function.
 * @param A Array A.
 * @param B Array B.
 * @param C Sum of array elements A and B directly across.
 * @param arraySize Size of arrays A, B, and C.
 */
int parallel_vec_add(int *A, int *B, int *C, int arraySize)
{

    // Initialize device pointers.
    int *d_A, *d_B, *d_C;

    // Allocate device memory.
    hipMalloc((void **)&d_A, arraySize * sizeof(int));
    hipMalloc((void **)&d_B, arraySize * sizeof(int));
    hipMalloc((void **)&d_C, arraySize * sizeof(int));

    // Transfer arrays a and b to device.
    hipMemcpy(d_A, A, arraySize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, arraySize * sizeof(int), hipMemcpyHostToDevice);

    // Calculate blocksize and gridsize.
    dim3 blockSize(512, 1, 1);
    dim3 gridSize(512 / arraySize + 1, 1);

    // Launch CUDA kernel.
    vectorAdditionKernel<<<gridSize, blockSize>>>(d_A, d_B, d_C, arraySize);

    // Copy result array c back to host memory.
    hipMemcpy(C, d_C, arraySize * sizeof(int), hipMemcpyDeviceToHost);

    // Free CUDA memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
